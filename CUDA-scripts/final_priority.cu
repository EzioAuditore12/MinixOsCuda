#include <chrono>
#include <climits>
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>
#include <ostream>
#include <string>
#include <sstream>
#include <vector>
#include <algorithm>

#define BLOCK_SIZE 256

__device__ int dev_time;
__device__ int dev_completed;

struct Process {
    int id;
    int arrival;
    int burst;
    int priority;
    int completion;
    int remaining;
    bool completed;
};

int get_process_size(std::string file_name) {
    std::string line;
    int process_count = 0;
    std::ifstream t_file(file_name);

    if (std::getline(t_file, line)) {
        std::istringstream iss(line);
        int value;
        process_count = 0;

        while(iss >> value) {
            process_count++;
        }
    }

    return process_count;
}

void file_reader_assigner(std::string process_file, int *at_array, int *bt_array, int *pt_array) {
    std::ifstream p_file(process_file);
    std::string line;
    int process_count = 0;

    if (std::getline(p_file, line)) {
        std::istringstream iss(line);
        int value;
        process_count = 0;

        while(iss >> value) {
            at_array[process_count++] = value;
        }
    }
    
    if (std::getline(p_file, line)) {
        std::istringstream iss(line);
        int value;
        int i = 0;
        while((iss >> value) && i < process_count) {
            bt_array[i++] = value;
        }
    }

    if (std::getline(p_file, line)) {
        std::istringstream iss(line);
        int value;
        int i = 0;
        while((iss >> value) && i < process_count) {
            pt_array[i++] = value;
        }
    }
    
    std::cout << "File size: " << process_count << std::endl;
}

// CPU Scheduler Function
void cpu_priority_scheduler(int N, int *arrival, int *burst, int *priority, 
                          int *is_completed, int *completion_time, int *remaining) {

    for (int i = 0; i < N; i++) {
        remaining[i] = burst[i];
        is_completed[i] = 0;
        completion_time[i] = 0;
    }

    int current_time = 0;
    int completed = 0;

    while (completed < N) {

        int highest_priority = INT_MAX;
        int selected_process = -1;

        for (int i = 0; i < N; i++) {
            if (!is_completed[i] && arrival[i] <= current_time && remaining[i] > 0) {
                if (priority[i] < highest_priority) {
                    highest_priority = priority[i];
                    selected_process = i;
                }
            }
        }

        if (selected_process == -1) {

            current_time++;
        } else {

            remaining[selected_process]--;
            
            // If process completes
            if (remaining[selected_process] == 0) {
                is_completed[selected_process] = 1;
                completion_time[selected_process] = current_time + 1;
                completed++;
            }
            
            current_time++;
        }
    }
}

// GPU Scheduler Kernel
__global__ void scheduler_kernel(int *arrival, int *remaining, int *priority, 
                                int *is_completed, int *completion_time, int N) {
    __shared__ unsigned int s_best_packed;  
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    while (true) {
        // reset block's best
        if (threadIdx.x == 0) s_best_packed = 0xFFFFFFFFu;
        __syncthreads();

        int cur_time = atomicAdd(&dev_time, 0);

        if (tid < N && !is_completed[tid]
            && arrival[tid] <= cur_time
            && remaining[tid] > 0) {
            unsigned int pack = ((unsigned int)priority[tid] << 16) | (unsigned int)tid;
            atomicMin(&s_best_packed, pack);
        }
        __syncthreads();

        if (threadIdx.x == 0) {
            unsigned int best = s_best_packed;
            if (best != 0xFFFFFFFFu) {
                int idx = best & 0xFFFF;
               
                int prev = atomicSub(&remaining[idx], 1);
                if (prev == 1) {
                    is_completed[idx] = 1;
                    completion_time[idx] = cur_time + 1;
                    atomicAdd(&dev_completed, 1);
                }
            }
            // Go to next time unit (like t0 -> t1)
            atomicAdd(&dev_time, 1);
        }
        __syncthreads();
       
        if (atomicAdd(&dev_completed, 0) >= N) break;
    }
}


void display_results(int N, int *arrival, int *burst, int *priority, 
                   int *is_completed, int *completion_time, int *remaining, int final_time, int final_done, bool display_processes) {
    std::cout << "Simulated time steps: " << final_time << ", Completed: " << final_done << "/" << N << "\n";

    if (!display_processes) {
        return; 
    }

    // Displaying the output of Initial process (read from file)
    std::cout << "Processes result: \n";
    
    std::cout << "\nPid\tAT\tBT\tPr\tComp\tRemaining\n";
    for(int i = 0; i < N; i++) {
        std::cout << i << "\t" << arrival[i] << "\t" << burst[i] << "\t" << priority[i] 
                  << "\t" << (is_completed[i] ? "Yes" : "No") << "\t" << remaining[i] << "\n";
    }

    std::vector<Process> processes(N);
    for (int i = 0; i < N; i++) {
        processes[i] = {
            i,            
            arrival[i],       
            burst[i],        
            priority[i],       
            completion_time[i],
            remaining[i],  
            (bool)is_completed[i] 
        };
    }
    
    // Sort processes by completion time
    std::sort(processes.begin(), processes.end(), 
              [](const Process& a, const Process& b) {
                  if (a.completed && b.completed) {
                      return a.completion < b.completion;
                  }
                  return a.completed > b.completed;
              });
    
    
    std::cout << "\nProcesses in scheduled order:\n";
    std::cout << "Pid\tAT\tBT\tPr\tCT\tTAT\tWT\n";
    for (const auto& p : processes) {
        if (p.completed) {
            int turnaround = p.completion - p.arrival;
            int waiting = turnaround - p.burst;
            std::cout << p.id << "\t" << p.arrival << "\t" << p.burst << "\t" 
                      << p.priority << "\t" << p.completion << "\t" 
                      << turnaround << "\t" << waiting << "\n";
        }
    }
}

int main(int argc, char *argv[]) {
    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " [-cpu|-gpu] <filename> [-display|-nodisplay]" << std::endl;
        return 1;
    }
    
    std::string mode = argv[1];
    std::string filename = argv[2];
    
    // Set default display mode
    bool display_processes = false;
    
    // Check for display flag
    if (argc >= 4) {
        std::string display_flag = argv[3];
        if (display_flag == "-display") {
            display_processes = true;
        } else if (display_flag == "-nodisplay") {
            display_processes = false;
        } else {
            std::cerr << "Invalid display flag. Use -display or -nodisplay." << std::endl;
            return 1;
        }
    }
    
    int N = get_process_size(filename);
    
    int *h_arrival = new int[N];
    int *h_burst = new int[N];
    int *h_prior = new int[N];
    int *h_completed = new int[N];
    int *h_completion = new int[N];
    int *h_remaining = new int[N];
    
    file_reader_assigner(filename, h_arrival, h_burst, h_prior);
    
    if (mode == "-cpu") {
        // CPU Mode
        std::cout << "Running CPU scheduler...\n";
        auto t0 = std::chrono::high_resolution_clock::now();
        cpu_priority_scheduler(N, h_arrival, h_burst, h_prior, h_completed, h_completion, h_remaining);
        auto t1 = std::chrono::high_resolution_clock::now();
        
        double cput = std::chrono::duration<double>(t1-t0).count();
        std::cout << "CPU time: " << cput << " s\n";
        
        // Count completed processes
        int final_done = 0;
        for (int i = 0; i < N; i++) {
            if (h_completed[i]) final_done++;
        }
        
        int final_time = 0;
        for (int i = 0; i < N; i++) {
            if (h_completed[i] && h_completion[i] > final_time) {
                final_time = h_completion[i];
            }
        }
        
        display_results(N, h_arrival, h_burst, h_prior, h_completed, h_completion, h_remaining, final_time, final_done, display_processes);
    }
    else if (mode == "-gpu") {
        // GPU Mode
        int *d_arr, *d_rem, *d_pri, *d_done, *d_completion;
        hipMalloc(&d_arr, N*sizeof(int));
        hipMalloc(&d_rem, N*sizeof(int));
        hipMalloc(&d_pri, N*sizeof(int));
        hipMalloc(&d_done, N*sizeof(int));
        hipMalloc(&d_completion, N*sizeof(int));
        
        hipMemcpy(d_arr, h_arrival, N*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_pri, h_prior, N*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_rem, h_burst, N*sizeof(int), hipMemcpyHostToDevice);
        hipMemset(d_done, 0, N*sizeof(int));
        hipMemset(d_completion, 0, N*sizeof(int));
        
        int zero = 0;
        hipMemcpyToSymbol(HIP_SYMBOL(dev_time), &zero, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(dev_completed), &zero, sizeof(int));
        
        dim3 block(BLOCK_SIZE), grid((N+BLOCK_SIZE-1)/BLOCK_SIZE);
        
        std::cout << "Running GPU scheduler...\n";
        auto t0 = std::chrono::high_resolution_clock::now();
        scheduler_kernel<<<grid,block>>>(d_arr, d_rem, d_pri, d_done, d_completion, N);
        hipDeviceSynchronize();
        auto t1 = std::chrono::high_resolution_clock::now();
        
        double gput = std::chrono::duration<double>(t1-t0).count();
        std::cout << "GPU time: " << gput << " s\n";
        
        int final_time = 0, final_done = 0;
        hipMemcpyFromSymbol(&final_time, HIP_SYMBOL(dev_time), sizeof(int));
        hipMemcpyFromSymbol(&final_done, HIP_SYMBOL(dev_completed), sizeof(int));
        
        // Copy device memory back to host only if we need to display results
        if (display_processes) {
            hipMemcpy(h_arrival, d_arr, N*sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(h_prior, d_pri, N*sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(h_completed, d_done, N*sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(h_remaining, d_rem, N*sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(h_completion, d_completion, N*sizeof(int), hipMemcpyDeviceToHost);
        }
        
        display_results(N, h_arrival, h_burst, h_prior, h_completed, h_completion, h_remaining, final_time, final_done, display_processes);
        
        // Free device memory
        hipFree(d_arr);
        hipFree(d_rem);
        hipFree(d_pri);
        hipFree(d_done);
        hipFree(d_completion);
    }
    else {
        std::cerr << "Invalid mode. Use -cpu or -gpu." << std::endl;
        return 1;
    }
    
    // Free host memory
    delete[] h_arrival;
    delete[] h_burst;
    delete[] h_prior;
    delete[] h_completed;
    delete[] h_completion;
    delete[] h_remaining;
    
    return 0;
}