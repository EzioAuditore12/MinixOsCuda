#include <climits>
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>
#include <ostream>
#include <string>



void process_file_setter(int n, int arrival_size, int burst_size, int priority_size, std::string path){
    // FILE *process_file = fopen("process_file.txt", "w");
    std::ofstream process_file(path);

    for(int i=0;i<n;i++) {
        process_file << (rand() % arrival_size) + 1 << " ";
    }
    process_file << std::endl;

    for(int i=0;i<n;i++) {
        process_file << (rand() % burst_size ) + 1 << " ";
    }
    process_file << std::endl;
    for(int i=0;i<n;i++) {
        process_file << (rand() % priority_size) + 1 << " ";
    }
     process_file << std::endl;
}


int main(int argc, char *argv[]){

    // std::cout<<"Lenght: "<<argc<<std::endl;

    // std::cout<<argv[1]<<" "<<argv[2]<<std::endl;

    // if(argv[1] == "-d"){
    //     // display the output
    // }

    // if(argv[1] == "-p"){
    //     // send the performance (do bigger)
    // }
    std::string path = argv[1];

    // std::string windows_desktop = "/mnt/c/Users/Manas Bisht/Desktop/";
    // std::string final_path = windows_desktop + path;
    std::string final_path = path;
    
    int process_num = atoi(argv[2]);

    std::cout<<"Final path: "<<final_path<<std::endl;
    process_file_setter(process_num, 100, 40, 20, final_path);
    
}